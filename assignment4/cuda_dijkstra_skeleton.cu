#include "hip/hip_runtime.h"
/* Name: Junxue ZHANG
 * ID: 20371613
 * Email: jzhangcs@ust.hk
 */

/*
 * This is code skeleton for COMP5112-17Spring assignment4
 * Compile: nvcc -std=c++11 -arch=sm_52 -o cuda_dijkstra cuda_dijkstra_skeleton.cu
 * Run: ./cuda_dijkstra -n <number of threads> -i <input file>,
 * you will find the output in 'output.txt' file
 *
 *  by Lipeng WANG, 5th Apr 2017
 */

#include <string>
#include <cassert>
#include <iostream>
#include <fstream>
#include <vector>
#include <climits>
#include <cstring>
#include <cmath>
#include <algorithm>
#include <sys/time.h>
#include <time.h>
#include <getopt.h>

#include <hip/hip_runtime.h>
#include <>

using std::string;
using std::cout;
using std::endl;
using std::vector;
using std::ceil;
using std::memcpy;

#define INF 1000000

/*
 * This is a CHECK function to check CUDA calls
 */
#define CHECK(call)                                                            \
 {                                                                              \
     const hipError_t error = call;                                            \
     if (error != hipSuccess)                                                  \
     {                                                                          \
         fprintf(stderr, "Error: %s:%d, ", __FILE__, __LINE__);                 \
         fprintf(stderr, "code: %d, reason: %s\n", error,                       \
                 hipGetErrorString(error));                                    \
         exit(1);                                                               \
     }                                                                          \
 }

/*
 * utils is a namespace for utility functions
 * including I/O (read input file and print results) and one matrix dimension convert(2D->1D) function
 */
namespace utils {
    int num_threads; //number of thread
    int N; //number of vertices
    int *mat; // the adjacency matrix

    string filename; // input file name
    string outputfile; //output file name, default: 'output.txt'

    void print_usage() {
        cout << "Usage:\n" << "\tcuda_dijkstra -n <number of threads per block> -i <input file>" << endl;
        exit(0);
    }

    int parse_args(int argc, char **argv) {
        filename = "";
        outputfile = "output.txt";
        num_threads = 0;

        int opt;
        if (argc < 2) {
            print_usage();
        }
        while ((opt = getopt(argc, argv, "n:i:o:h")) != EOF) {
            switch (opt) {
                case 'n':
                    num_threads = atoi(optarg);
                    break;
                case 'i':
                    filename = optarg;
                    break;
                case 'o':
                    outputfile = optarg;
                    break;
                case 'h':
                case '?':
                default:
                    print_usage();
            }
        }
        if (filename.length() == 0 || num_threads == 0)
            print_usage();
        return 0;
    }

    /*
     * convert 2-dimension coordinate to 1-dimension
     */
    int convert_dimension_2D_1D(int x, int y) {
        return x * N + y;
    }

    int read_file(string filename) {
        std::ifstream inputf(filename, std::ifstream::in);
        inputf >> N;
        assert(N < (1024 * 1024 *
                    20)); // input matrix should be smaller than 20MB * 20MB (400MB, we don't have too much memory for multi-processors)
        mat = (int *) malloc(N * N * sizeof(int));
        for (int i = 0; i < N; i++)
            for (int j = 0; j < N; j++) {
                inputf >> mat[convert_dimension_2D_1D(i, j)];
            }

        return 0;
    }

    string format_path(int i, int *pred) {
        string out("");
        int current_vertex = i;
        while (current_vertex != 0) {
            string s = std::to_string(current_vertex);
            std::reverse(s.begin(), s.end());
            out = out + s + ">-";
            current_vertex = pred[current_vertex];
        }
        out = out + std::to_string(0);
        std::reverse(out.begin(), out.end());
        return out;
    }

    int print_result(int *dist, int *pred) {
        std::ofstream outputf(outputfile, std::ofstream::out);
        outputf << dist[0];
        for (int i = 1; i < N; i++) {
            outputf << " " << dist[i];
        }
        for (int i = 0; i < N; i++) {
            outputf << "\n";
            if (dist[i] >= 1000000) {
                outputf << "NO PATH";
            } else {
                outputf << format_path(i, pred);
            }
        }
        outputf << endl;
        return 0;
    }
}//namespace utils


//------You may add helper functions and global variables here------

/*
 * function: find the local minimum for each block and store them to d_local_min and d_local_min_index
 * parameters: N: input size, *d_visit: array to record which vertex has been visited, *d_all_dist: array to store the distance,
 *        *d_local_min: array to store the local minimum value for each block, *d_local_min_index: array to store the local minimum index for each block
 */
__global__ void
 FindLocalMin(int N, int *d_visit, int *d_all_dist, int *d_local_min, int *d_local_min_index) {
    const int bid = blockIdx.x;
    const int local_tid = threadIdx.x;
    const int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int local_nthread = blockDim.x;
    const int global_nthread = blockDim.x * gridDim.x;

    __shared__ int thread_level_mininum[1024];
    __shared__ int thread_level_mininum_index[1024];

    thread_level_mininum[local_tid] = INF;
    thread_level_mininum_index[local_tid] = -1;    

    for (int i = global_tid; i < N; i += global_nthread) {
    //  printf ("Thread %i (local %i in block %i) will process vertex %i\n", global_tid, local_tid, bid, i);
        if (!d_visit[i]) {
            if (d_all_dist[i] < thread_level_mininum[local_tid]) {
                thread_level_mininum[local_tid] = d_all_dist[i];
                thread_level_mininum_index[local_tid] = i;
            }
        }
    }

    __syncthreads();
    if (local_tid == 0) {
        d_local_min[bid] = INF;
        d_local_min_index[bid] = -1; 
        for (int j = 0; j < local_nthread; ++j) {
            if (thread_level_mininum[j] < d_local_min[bid]) {
                d_local_min[bid] = thread_level_mininum[j];
                d_local_min_index[bid] = thread_level_mininum_index[j];
            }
        }
    //  printf ("Local mininum for block %i is %i with vertex  %i\n", bid, d_local_min[bid], d_local_min_index[bid]);
    } 
}

/*
 * function: update the global minimum value(and index), store them to a global memory address
 * parameters: *global_min: memory address to store the global min value, *global_min_index: memory address to store the global min index
 *        *d_local_min: array stores the local min value od each block, *d_local_min_index: array stores the local min index of each block
 *        *d_visit: array stores the status(visited/un-visited) for each vertex
 */
__global__ void
UpdateGlobalMin(int *global_min, int *global_min_index, int *d_local_min, int *d_local_min_index, int *d_visit) {
    const int bid = blockIdx.x;
    const int tid = threadIdx.x;
    const int nblocks = gridDim.x;    

    if (bid == 0 && tid == 0) {
        *global_min = INF;
        *global_min_index = -1;
  
        for (int i = 0; i < nblocks; ++i) {
            if (d_local_min[i] < *global_min) {
                *global_min = d_local_min[i];
                *global_min_index = d_local_min_index[i];
            }          
        }
        d_visit[*global_min_index] = true;
    }
}

/*
 * function: update the shortest path for every un-visited vertices
 * parameters: N: input size, *mat: input matrix, *d_visit: array stores the status(visited/un-visited) for each vertex
 *             *d_all_dist: array stores the shortest distance for each vertex, *d_all_pred: array stores the predecessors
 *             *global_min: memory address that stores the global min value, *global_min_index: memory address that stores the global min index
 */
__global__ void
UpdatePath(int N, int *mat, int *d_visit, int *d_all_dist, int *d_all_pred, int *global_min, int *global_min_index) {
    const int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
    const int global_nthread = blockDim.x * gridDim.x;

    for (int i = global_tid; i < N; i += global_nthread) {
        if (!d_visit[i]) {
            const int new_dist = *global_min + mat[*global_min_index * N + i];
            if (new_dist < d_all_dist[i]) {
               d_all_dist[i] = new_dist;
               d_all_pred[i] = *global_min_index;
            }
        }
    }
}

//Do not change anything below this line
void dijkstra(int N, int p, int *mat, int *all_dist, int *all_pred) {

    //threads number for each block should smaller than or equal to 1024
    assert(p <= 1024);

    //we restrict this value to 8, DO NOT change it!
    int blocksPerGrid = 8;

    //NOTICE: (p * 8) may LESS THAN N
    int threadsPerBlock = p;

    dim3 blocks(blocksPerGrid);
    dim3 threads(threadsPerBlock);


    //allocate memory
    int *h_visit;
    int *d_mat, *d_visit, *d_all_dist, *d_all_pred, *d_local_min, *d_local_min_index;
    int *d_global_min, *d_global_min_index;

    h_visit = (int *) calloc(N, sizeof(int));
    hipMalloc(&d_mat, sizeof(int) * N * N);
    hipMalloc(&d_visit, sizeof(int) * N);
    hipMalloc(&d_all_dist, sizeof(int) * N);
    hipMalloc(&d_all_pred, sizeof(int) * N);
    hipMalloc(&d_local_min, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_local_min_index, sizeof(int) * blocksPerGrid);
    hipMalloc(&d_global_min, sizeof(int));
    hipMalloc(&d_global_min_index, sizeof(int));

    //initialization and copy data from host to device
    for (int i = 0; i < N; i++) {
        all_dist[i] = mat[i];
        all_pred[i] = 0;
        h_visit[i] = 0;
    }
    h_visit[0] = 1;

    hipMemcpy(d_mat, mat, sizeof(int) * N * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_dist, all_dist, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_all_pred, all_pred, sizeof(int) * N, hipMemcpyHostToDevice);
    hipMemcpy(d_visit, h_visit, sizeof(int) * N, hipMemcpyHostToDevice);

    //dijkstra iterations
    for (int iter = 1; iter < N; iter++) {
        FindLocalMin <<< blocks, threads >>> (N, d_visit, d_all_dist, d_local_min, d_local_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
        UpdateGlobalMin <<< blocks, threads >>>
                                    (d_global_min, d_global_min_index, d_local_min, d_local_min_index, d_visit);
        //CHECK(hipDeviceSynchronize()); //only for debug
        UpdatePath << < blocks, threads >> >
                                (N, d_mat, d_visit, d_all_dist, d_all_pred, d_global_min, d_global_min_index);
        //CHECK(hipDeviceSynchronize()); //only for debug
    }

    //copy results from device to host
    hipMemcpy(all_dist, d_all_dist, sizeof(int) * N, hipMemcpyDeviceToHost);
    hipMemcpy(all_pred, d_all_pred, sizeof(int) * N, hipMemcpyDeviceToHost);

    //free memory
    free(h_visit);
    hipFree(d_mat);
    hipFree(d_visit);
    hipFree(d_all_dist);
    hipFree(d_all_pred);
    hipFree(d_local_min);
    hipFree(d_local_min_index);
    hipFree(d_global_min);
    hipFree(d_global_min_index);

}

int main(int argc, char **argv) {
    assert(utils::parse_args(argc, argv) == 0);
    assert(utils::read_file(utils::filename) == 0);

    //`all_dist` stores the distances and `all_pred` stores the predecessors
    int *all_dist;
    int *all_pred;
    all_dist = (int *) calloc(utils::N, sizeof(int));
    all_pred = (int *) calloc(utils::N, sizeof(int));

    //time counter
    timeval start_wall_time_t, end_wall_time_t;
    float ms_wall;

    hipDeviceReset();

    //start timer
    gettimeofday(&start_wall_time_t, nullptr);
    dijkstra(utils::N, utils::num_threads, utils::mat, all_dist, all_pred);
    CHECK(hipDeviceSynchronize());

    //end timer
    gettimeofday(&end_wall_time_t, nullptr);
    ms_wall = ((end_wall_time_t.tv_sec - start_wall_time_t.tv_sec) * 1000 * 1000
               + end_wall_time_t.tv_usec - start_wall_time_t.tv_usec) / 1000.0;

    std::cerr << "Time(ms): " << ms_wall << endl;

    utils::print_result(all_dist, all_pred);

    free(utils::mat);
    free(all_dist);
    free(all_pred);

    return 0;
}
